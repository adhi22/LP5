#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vectorAdd(int *a, int *b, int *res, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < n) {
        res[index] = a[index] + b[index];
    }
}

int main() {
    int size = 10;
    
    int *arr1_cpu = new int[size];
    int *arr2_cpu = new int[size];
    int *res_cpu = new int[size];

    for(int i=0; i<size; i++) {
        arr1_cpu[i] = rand() % 100;
        arr2_cpu[i] = rand() % 100;
    }

    int *arr1_gpu, *arr2_gpu, *res_gpu;
    hipMalloc(&arr1_gpu, size * sizeof(int));
    hipMalloc(&arr2_gpu, size * sizeof(int));
    hipMalloc(&res_gpu, size * sizeof(int));

    // copy vectors from cpu to gpu
    hipMemcpy(arr1_gpu, arr1_cpu, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2_cpu, size * sizeof(int), cudaMemcpyToHostDevice);

    // vector addition on GPU
    int block_size = 256;
    int grid_size = (size + block_size - 1) / block_size;

    vectorAdd<<<grid_size, block_size>>>(arr1_gpu, arr2_gpu, res_gpu, size);

    // copy res to cpu
    cudeMemcpy(res_cpu, res_gpu, size * sizeof(int), hipMemcpyDeviceToHost);

    cout << "GPU Result : \n";
    for(int i=0; i<size; i++) cout << res_cpu[i] << " ";

    // vector add on cpu
    for(int i=0; i<size; i++) {
        res_cpu[i] = arr1_cpu[i] + arr2_cpu[i];
    }

    cout << "CPU Result : \n";
    for(int i=0; i<size; i++) cout << res_cpu[i] << " ";

    hipFree(arr1_gpu);
    hipFree(arr2_gpu);
    hipFree(res_gpu);

    delete[] arr1_cpu;
    delete[] arr2_cpu;
    delete[] res_cpu;

}